#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/video_3d_layers.hpp"

namespace caffe {

	template <typename Dtype>
	__global__ void StretchForward(const int nthreads, const Dtype* bottom_data,
		const int channels, const int offset, Dtype* top_data) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int o = index % offset;
			int c = index / offset;
			top_data[o * channels + c] =
				bottom_data[c * offset + o];
		}
	}

	template <typename Dtype>
	Dtype Stretch3DLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		vector<Blob<Dtype>*>* top) {
		Dtype* top_data = (*top)[0]->mutable_gpu_data();
		const Dtype* bottom_data = bottom[0]->gpu_data();
		int count = count_ / num_;
		int offset = length_ * height_ * width_;
		for (int n = 0; n < num_; ++n)
		{
			StretchForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
				count, bottom_data + bottom[0]->offset(n), channels_, offset ,
				top_data + (*top)[0]->offset(n * offset));
		}
		return Dtype(0.);
	}

	template <typename Dtype>
	__global__ void StretchBackward(const int nthreads, const Dtype* top_diff,
		const int channels, const int offset, Dtype* bottom_diff) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			int o = index % offset;
			int c = index / offset;
			bottom_diff[c * offset + o] =
				top_diff[o * channels + c];
		}
	}

	template <typename Dtype>
	void Stretch3DLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
		const Dtype* top_diff = top[0]->gpu_diff();
		Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
		int count = count_ / num_;
		int offset = length_ * width_ * height_;
		
		if (!propagate_down) { return; }
		for (int n = 0; n < num_; ++n)
		{
			StretchBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
				count, top_diff + top[0]->offset(n * offset), channels_, offset,
				bottom_diff + (*bottom)[0]->offset(n));
		}
	}

	INSTANTIATE_CLASS(Stretch3DLayer);
}  // namespace caffe
