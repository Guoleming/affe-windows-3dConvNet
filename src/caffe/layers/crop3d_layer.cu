#include "hip/hip_runtime.h"
/*
 *
 *  Copyright (c) 2015, Facebook, Inc. All rights reserved.
 *
 *  Licensed under the Creative Commons Attribution-NonCommercial 3.0
 *  License (the "License"). You may obtain a copy of the License at
 *  https://creativecommons.org/licenses/by-nc/3.0/.
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  License for the specific language governing permissions and limitations
 *  under the License.
 *
 *
 */

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/video_3d_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;
using std::min;

namespace caffe {
	
  // Copy (one line per thread) from one array to another, with arbitrary 
 // strides in the last two dimensions. 
 template <typename Dtype> 
 __global__ void copy_kernel(const int n, const int height, const int width, 
     const int src_outer_stride, const int src_inner_stride, 
     const int dest_outer_stride, const int dest_inner_stride, 
     const Dtype* src, Dtype* dest) { 
   CUDA_KERNEL_LOOP(index, n) { 
     int src_start = index / height * src_outer_stride 
                   + index % height * src_inner_stride; 
     int dest_start = index / height * dest_outer_stride 
                    + index % height * dest_inner_stride; 
     for (int i = 0; i < width; ++i) { 
       dest[dest_start + i] = src[src_start + i]; 
     } 
   } 
 } 

 template <typename Dtype> 
 __global__ void copy_kernel_3d(const int n, const int length, const int height, const int width,
	 const int src_stride1, const int src_stride2, const int src_stride3,
	 const int dest_stride1, const int dest_stride2, const int dest_stride3,
	 const Dtype* src, Dtype* dest) {
	 CUDA_KERNEL_LOOP(index, n) {
		 int src_start = index % height * src_stride3 + index / height % length * src_stride2 + index / (height * length) * src_stride1;
		 int dest_start = index % height * dest_stride3 + index / height % length * dest_stride2 + index / (height * length) * dest_stride1;

		 for (int i = 0; i < width; ++i) {
			 dest[dest_start + i] = src[src_start + i];
		 }
	 }
 }

template <typename Dtype>
Dtype Crop3DLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
   const Dtype* bottom_data = bottom[0]->gpu_data(); 
   Dtype* top_data = (*top)[0]->mutable_gpu_data(); 
   const int lines = (*top)[0]->count() / (*top)[0]->width();

   // NOLINT_NEXT_LINE(whitespace/operators) 
   copy_kernel_3d << <CAFFE_GET_BLOCKS(lines), CAFFE_CUDA_NUM_THREADS >> >(
	   lines, (*top)[0]->length(), (*top)[0]->height(), (*top)[0]->width(),
       bottom[0]->length() * bottom[0]->height() * bottom[0]->width(), bottom[0]->height() * bottom[0]->width(), bottom[0]->width(), 
	   (*top)[0]->length() * (*top)[0]->height() * (*top)[0]->width(), (*top)[0]->height() * (*top)[0]->width(), (*top)[0]->width(),
       bottom_data + bottom[0]->offset(0, 0, crop_l_, crop_h_, crop_w_), top_data); 

  return Dtype(0.);
}

template <typename Dtype>
void Crop3DLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff(); 
   Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff(); 
   const int lines = top[0]->count() / top[0]->width(); 
 
   if (propagate_down) { 
	   caffe_gpu_set((*bottom)[0]->count(), static_cast<Dtype>(0), bottom_diff);
	   caffe_gpu_set((*bottom)[1]->count(), static_cast<Dtype>(0), (*bottom)[1]->mutable_gpu_diff());
     // NOLINT_NEXT_LINE(whitespace/operators) 
	   copy_kernel_3d << <CAFFE_GET_BLOCKS(lines), CAFFE_CUDA_NUM_THREADS >> >(
         lines, top[0]->length(), top[0]->height(), top[0]->width(), 
         top[0]->length() * top[0]->height() * top[0]->width(), top[0]->height() * top[0]->width(), top[0]->width(), 
		 (*bottom)[0]->length() * (*bottom)[0]->height() * (*bottom)[0]->width(), (*bottom)[0]->height() * (*bottom)[0]->width(), (*bottom)[0]->width(),
		 top_diff, bottom_diff + (*bottom)[0]->offset(0, 0, crop_l_, crop_h_, crop_w_));
   } 

}

INSTANTIATE_CLASS(Crop3DLayer);

}  // namespace caffe
